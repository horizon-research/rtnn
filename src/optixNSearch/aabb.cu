#include "hip/hip_runtime.h"
#include <optix.h>
#include <sutil/vec_math.h>

__global__ void kGenAABB_t (
      const float3* points,
      float radius,
      float* radii,
      unsigned int N,
      OptixAabb* aabb
)
{
  unsigned int particleIndex = blockIdx.x * blockDim.x + threadIdx.x;
  if (particleIndex >= N) return;

  float3 center = points[particleIndex];
  float actual_radius = radii == NULL ? radius : radii[particleIndex];

  float3 m_min = center - actual_radius;
  float3 m_max = center + actual_radius;

  aabb[particleIndex] =
  {
    m_min.x, m_min.y, m_min.z,
    m_max.x, m_max.y, m_max.z
  };
}

void kGenAABB(
  float3* points, 
  float radius,
  float* radii, 
  unsigned int numPrims, 
  OptixAabb* d_aabb, 
  hipStream_t stream
) {
  unsigned int threadsPerBlock = 64;
  unsigned int numOfBlocks = numPrims / threadsPerBlock + 1;

  kGenAABB_t <<<numOfBlocks, threadsPerBlock, 0, stream>>> (
      points,
      radius,
      radii,
      numPrims,
      d_aabb
     );
}
